#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include "hist-equ.h"


void histogram(int * hist_out, unsigned char * img_in, int img_size, int nbr_bin) {
    int i;

    // Initialization
    for (i = 0; i < nbr_bin; i++) {
        hist_out[i] = 0;
    }

    // Constructs the Histogram Vector
    for (i = 0; i < img_size; i++) {
        hist_out[img_in[i]]++;
    }
}

void histogram_equalization(unsigned char * img_out, unsigned char * img_in, 
                            int * hist_in, int img_size, int nbr_bin) {
    int *lut = (int *)malloc(sizeof(int)*nbr_bin);
    int i, cdf, min, d, index;

    /* Construct the LUT by calculating the CDF */
    cdf = 0;
    min = 0;
    i = 0;
    
    // Finds the first value on the Histogram that isn't 0
    while(min == 0) {
        min = hist_in[i++];
    }
    index = i-1;

    // Calculate the look up table (lut)
    for (i = 0; i < index + 1; i++)  {
        lut[i] = 0;
    }

    d = img_size - min;
    for(i = index; i < nbr_bin; i++) {
        cdf += hist_in[i];
        lut[i] = (int)(((float)cdf - min)*255/d + 0.5);
    }

    for(i = 0; i < nbr_bin; i++)  {
       if(lut[i] > 255) {
            lut[i] = 255;
        }
    }                          
    
    /* Get the result image */
    for(i = 0; i < img_size; i++) {
        img_out[i] = (unsigned char)lut[img_in[i]];
    }
}

__global__ void histogramGPU(int * hist_out) {
    int index = threadIdx.x;

    // Initialization
    hist_out[index] = 0;

    // Constructs the Histogram Vector
    /*for (i = 0; i < img_size; i++) {
        hist_out[img_in[i]]++;
    }*/
}
__global__ void histogramConstuctionGPU(int * hist_out, unsigned char * img_in, int imageW, int imageH) {
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    int y = index / imageW; // row
    int x = index % imageW; // col

    // Constructs the Histogram Vector
    hist_out[img_in[y * imageW + x]]++;
    __syncthreads();
}

/*__global__ void convolutionRowGPU(float *d_Dst, float *d_Src, float *d_Filter, int imageW, int imageH, int filterR) {

  int index = blockIdx.x*blockDim.x + threadIdx.x;
  int y = index / imageW; // row
  int x = index % imageW; // col
  y += filterR;
  x += filterR;
  float sum = 0;
    for(int k = -filterR; k <= filterR; k++){
      int d = x + k;
        sum += d_Src[y * (imageW+2*filterR) + d] * d_Filter[filterR - k];
    }
    d_Dst[y * (imageW + 2 * filterR) + x] = sum; 
}*/