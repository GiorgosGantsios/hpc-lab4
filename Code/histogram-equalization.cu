#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include "hist-equ.h"

extern int error_flag;

void histogram(int * hist_out, unsigned char * img_in, int img_size, int nbr_bin) {
    int i;

    // Initialization
    for (i = 0; i < nbr_bin; i++) {
        hist_out[i] = 0;
    }

    // Constructs the Histogram Vector
    for (i = 0; i < img_size; i++) {
        hist_out[img_in[i]]++;
    }
}

void histogram_equalization(unsigned char * img_out, unsigned char * img_in, 
                            int * hist_in, int img_size, int nbr_bin) {
    int *lut = (int *)malloc(sizeof(int)*nbr_bin);
    if(!lut){
        error_flag = 1;
        return;
    }
    int i, cdf, min, d, index;

    /* Construct the LUT by calculating the CDF */
    cdf = 0;
    min = 0;
    i = 0;

    // Finds the first value on the Histogram that isn't 0
    while(min == 0) {
        min = hist_in[i++];
    }
    index = i-1;

    // Calculate the look up table (lut)
    for (i = 0; i < index + 1; i++)  {
        lut[i] = 0;
    }

    d = img_size - min;
    for(i = index; i < nbr_bin; i++) {
        cdf += hist_in[i];
        lut[i] = (int)(((float)cdf - min)*255/d + 0.5);
    }

    for(i = 0; i < nbr_bin; i++)  {
       if(lut[i] > 255) {
            lut[i] = 255;
        }
    }                          
    
    /* Get the result image */
    for(i = 0; i < img_size; i++) {
        img_out[i] = (unsigned char)lut[img_in[i]];
    }
}