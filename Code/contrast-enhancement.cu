#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include "hist-equ.h"

PGM_IMG contrast_enhancement_g(PGM_IMG img_in)  {
    PGM_IMG result;
    int hist[256];

    result.w = img_in.w;
    result.h = img_in.h;
    result.img = (unsigned char *)malloc(result.w * result.h * sizeof(unsigned char));

    histogram(hist, img_in.img, img_in.h * img_in.w, 256);
    histogram_equalization(result.img,img_in.img,hist,result.w*result.h, 256);

    return result;
}

PGM_IMG contrast_enhancement_GPU(PGM_IMG img_in)  {
    hipEvent_t startCuda, stopCuda;
    PGM_IMG gpuResult;
    PGM_IMG result;
    float millisecondsTransfers = 0, time;
    //int hist[256];
    int t_hist[256];
    int *d_hist, *A0_hist, *A1_hist, *A2_hist, *A3_hist, *A4_hist, *A5_hist, *A6_hist, *A7_hist;
    unsigned char * d_ImgIn;
    unsigned char * img_A0,* img_A1,* img_A2,* img_A3,* img_A4,* img_A5,* img_A6,* img_A7;
    int SegSize = 114688;
    int t0_hist[256], t1_hist[256], t2_hist[256], t3_hist[256], t4_hist[256], t5_hist[256], t6_hist[256], t7_hist[256];
    hipStream_t stream0, stream1, stream2, stream3, stream4, stream5, stream6, stream7;
    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);
    // hipStreamCreate(&stream2);
    // hipStreamCreate(&stream3);
    // hipStreamCreate(&stream4);
    // hipStreamCreate(&stream5);
    // hipStreamCreate(&stream6);
    // hipStreamCreate(&stream7);

    result.w = img_in.w;
    result.h = img_in.h;
    result.img = (unsigned char *)malloc(result.w * result.h * sizeof(unsigned char));

    gpuResult.w = img_in.w;
    gpuResult.h = img_in.h;

    hipEventCreate(&startCuda);
    hipEventCreate(&stopCuda);

    hipEventRecord(startCuda);


    hipError_t err = hipMalloc((void **)&gpuResult.img, gpuResult.w * gpuResult.h * sizeof(unsigned char));
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA malloc error: %s\n", hipGetErrorString(err));
        if (gpuResult.img) hipFree(gpuResult.img);
        return(gpuResult);
    }
    err = hipMalloc((void **)&d_ImgIn, gpuResult.w * gpuResult.h * sizeof(unsigned char));
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA malloc error: %s\n", hipGetErrorString(err));
        if (gpuResult.img) hipFree(gpuResult.img);
        return(gpuResult);
    }
    err = hipMalloc((void**)&d_hist, 256 * sizeof(int));  // Allocate memory on the GPU
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA malloc error: %s\n", hipGetErrorString(err));
        if (gpuResult.img) hipFree(gpuResult.img);
        return(gpuResult);
    }
    // err = hipMalloc((void**)&A0_hist, 256 * sizeof(int));  // Allocate memory on the GPU
    // if (err != hipSuccess) {
    //     fprintf(stderr, "CUDA malloc error: %s\n", hipGetErrorString(err));
    //     if (gpuResult.img) hipFree(gpuResult.img);
    //     return(gpuResult);
    // }
    // err = hipMalloc((void**)&A1_hist, 256 * sizeof(int));  // Allocate memory on the GPU
    // if (err != hipSuccess) {
    //     fprintf(stderr, "CUDA malloc error: %s\n", hipGetErrorString(err));
    //     if (gpuResult.img) hipFree(gpuResult.img);
    //     return(gpuResult);
    // }
    // err = hipMalloc((void**)&A2_hist, 256 * sizeof(int));  // Allocate memory on the GPU
    // if (err != hipSuccess) {
    //     fprintf(stderr, "CUDA malloc error: %s\n", hipGetErrorString(err));
    //     if (gpuResult.img) hipFree(gpuResult.img);
    //     return(gpuResult);
    // }
    // err = hipMalloc((void**)&A3_hist, 256 * sizeof(int));  // Allocate memory on the GPU
    // if (err != hipSuccess) {
    //     fprintf(stderr, "CUDA malloc error: %s\n", hipGetErrorString(err));
    //     if (gpuResult.img) hipFree(gpuResult.img);
    //     return(gpuResult);
    // }
    // err = hipMalloc((void**)&A4_hist, 256 * sizeof(int));  // Allocate memory on the GPU
    // if (err != hipSuccess) {
    //     fprintf(stderr, "CUDA malloc error: %s\n", hipGetErrorString(err));
    //     if (gpuResult.img) hipFree(gpuResult.img);
    //     return(gpuResult);
    // }
    // err = hipMalloc((void**)&A5_hist, 256 * sizeof(int));  // Allocate memory on the GPU
    // if (err != hipSuccess) {
    //     fprintf(stderr, "CUDA malloc error: %s\n", hipGetErrorString(err));
    //     if (gpuResult.img) hipFree(gpuResult.img);
    //     return(gpuResult);
    // }
    // err = hipMalloc((void**)&A6_hist, 256 * sizeof(int));  // Allocate memory on the GPU
    // if (err != hipSuccess) {
    //     fprintf(stderr, "CUDA malloc error: %s\n", hipGetErrorString(err));
    //     if (gpuResult.img) hipFree(gpuResult.img);
    //     return(gpuResult);
    // }
    // err = hipMalloc((void**)&A7_hist, 256 * sizeof(int));  // Allocate memory on the GPU
    // if (err != hipSuccess) {
    //     fprintf(stderr, "CUDA malloc error: %s\n", hipGetErrorString(err));
    //     if (gpuResult.img) hipFree(gpuResult.img);
    //     return(gpuResult);
    // }
    err = hipMalloc((void**)&img_A0, SegSize * sizeof(unsigned char));  // Allocate memory on the GPU
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA malloc error: %s\n", hipGetErrorString(err));
        if (gpuResult.img) hipFree(gpuResult.img);
        return(gpuResult);
    }
    err = hipMalloc((void**)&img_A1, SegSize * sizeof(unsigned char));  // Allocate memory on the GPU
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA malloc error: %s\n", hipGetErrorString(err));
        if (gpuResult.img) hipFree(gpuResult.img);
        return(gpuResult);
    }
    // err = hipMalloc((void**)&img_A2, SegSize * sizeof(unsigned char));  // Allocate memory on the GPU
    // if (err != hipSuccess) {
    //     fprintf(stderr, "CUDA malloc error: %s\n", hipGetErrorString(err));
    //     if (gpuResult.img) hipFree(gpuResult.img);
    //     return(gpuResult);
    // }
    // err = hipMalloc((void**)&img_A3, SegSize * sizeof(unsigned char));  // Allocate memory on the GPU
    // if (err != hipSuccess) {
    //     fprintf(stderr, "CUDA malloc error: %s\n", hipGetErrorString(err));
    //     if (gpuResult.img) hipFree(gpuResult.img);
    //     return(gpuResult);
    // }
    // err = hipMalloc((void**)&img_A4, SegSize * sizeof(unsigned char));  // Allocate memory on the GPU
    // if (err != hipSuccess) {
    //     fprintf(stderr, "CUDA malloc error: %s\n", hipGetErrorString(err));
    //     if (gpuResult.img) hipFree(gpuResult.img);
    //     return(gpuResult);
    // }
    // err = hipMalloc((void**)&img_A5, SegSize * sizeof(unsigned char));  // Allocate memory on the GPU
    // if (err != hipSuccess) {
    //     fprintf(stderr, "CUDA malloc error: %s\n", hipGetErrorString(err));
    //     if (gpuResult.img) hipFree(gpuResult.img);
    //     return(gpuResult);
    // }
    // err = hipMalloc((void**)&img_A6, SegSize * sizeof(unsigned char));  // Allocate memory on the GPU
    // if (err != hipSuccess) {
    //     fprintf(stderr, "CUDA malloc error: %s\n", hipGetErrorString(err));
    //     if (gpuResult.img) hipFree(gpuResult.img);
    //     return(gpuResult);
    // }
    // err = hipMalloc((void**)&img_A7, SegSize * sizeof(unsigned char));  // Allocate memory on the GPU
    // if (err != hipSuccess) {
    //     fprintf(stderr, "CUDA malloc error: %s\n", hipGetErrorString(err));
    //     if (gpuResult.img) hipFree(gpuResult.img);
    //     return(gpuResult);
    // }
    
    hipMemset(d_hist, 0, sizeof(int) * 256);

    err = hipMemcpy(d_ImgIn, img_in.img, gpuResult.w * gpuResult.h * sizeof(unsigned char), hipMemcpyHostToDevice);  // Copy data from host to device
    //err = hipMemcpy(img_A0, img_in.img, SegSize * sizeof(unsigned char), hipMemcpyHostToDevice);  // Copy data from host to device

    //hipMemcpyAsync(img_A0, d_ImgIn, SegSize*sizeof(unsigned char), hipMemcpyHostToDevice,  stream0);
    //histogramGPU<<<((SegSize)/256), 256, 256*sizeof(int), stream0 >>>(A0_hist, img_A0, SegSize);
    //histogramGPU<<<((gpuResult.h*gpuResult.w)/256)+1, 256, 256*sizeof(int) >>>(d_hist, d_ImgIn, gpuResult.w* gpuResult.h);

    for (int i=0; i<gpuResult.h*gpuResult.w; i+=SegSize*2)  {
        hipMemcpyAsync(img_A0, img_in.img+i, SegSize*sizeof(unsigned char), hipMemcpyHostToDevice,  stream0);
        hipMemcpyAsync(img_A1, img_in.img+i+SegSize, SegSize*sizeof(unsigned char), hipMemcpyHostToDevice, stream1); 
        // hipMemcpyAsync(img_A2, img_in.img+i+2 * SegSize, SegSize*sizeof(unsigned char), hipMemcpyHostToDevice, stream2);
        // hipMemcpyAsync(img_A3, img_in.img+i+3 * SegSize, SegSize*sizeof(unsigned char), hipMemcpyHostToDevice, stream3);
        // hipMemcpyAsync(img_A4, img_in.img+i+4 * SegSize, SegSize*sizeof(unsigned char), hipMemcpyHostToDevice, stream4);
        // hipMemcpyAsync(img_A5, img_in.img+i+5 * SegSize, SegSize*sizeof(unsigned char), hipMemcpyHostToDevice, stream5);
        // hipMemcpyAsync(img_A6, img_in.img+i+6 * SegSize, SegSize*sizeof(unsigned char), hipMemcpyHostToDevice, stream6);
        // hipMemcpyAsync(img_A7, img_in.img+i+7 * SegSize, SegSize*sizeof(unsigned char), hipMemcpyHostToDevice, stream7);
        histogramGPU<<<((SegSize)/256), 256, 256*sizeof(int), stream0 >>>(d_hist, img_A0, SegSize);
        histogramGPU<<<((SegSize)/256), 256, 256*sizeof(int), stream1 >>>(d_hist, img_A1, SegSize);
        // histogramGPU<<<((SegSize)/256), 256, 256*sizeof(int), stream2 >>>(d_hist, img_A2, SegSize);
        // histogramGPU<<<((SegSize)/256), 256, 256*sizeof(int), stream3 >>>(d_hist, img_A3, SegSize);
        // histogramGPU<<<((SegSize)/256), 256, 256*sizeof(int), stream4 >>>(d_hist, img_A4, SegSize);
        // histogramGPU<<<((SegSize)/256), 256, 256*sizeof(int), stream5 >>>(d_hist, img_A5, SegSize);
        // histogramGPU<<<((SegSize)/256), 256, 256*sizeof(int), stream6 >>>(d_hist, img_A6, SegSize);
        // histogramGPU<<<((SegSize)/256), 256, 256*sizeof(int), stream7 >>>(d_hist, img_A7, SegSize);
        // hipMemcpyAsync(t0_hist, A0_hist, 256*sizeof(int), hipMemcpyDeviceToHost, stream0);
        // hipMemcpyAsync(t1_hist, A1_hist, 256*sizeof(int), hipMemcpyDeviceToHost, stream1);
        // hipMemcpyAsync(t2_hist, A2_hist, 256*sizeof(int), hipMemcpyDeviceToHost, stream2);
        // hipMemcpyAsync(t3_hist, A3_hist, 256*sizeof(int), hipMemcpyDeviceToHost, stream3);
        // hipMemcpyAsync(t4_hist, A4_hist, 256*sizeof(int), hipMemcpyDeviceToHost, stream4);
        // hipMemcpyAsync(t5_hist, A5_hist, 256*sizeof(int), hipMemcpyDeviceToHost, stream5);
        // hipMemcpyAsync(t6_hist, A6_hist, 256*sizeof(int), hipMemcpyDeviceToHost, stream6);
        // hipMemcpyAsync(t7_hist, A7_hist, 256*sizeof(int), hipMemcpyDeviceToHost, stream7);
        hipStreamSynchronize(stream0);
        hipStreamSynchronize(stream1);
        // hipStreamSynchronize(stream2);
        // hipStreamSynchronize(stream3);
        // hipStreamSynchronize(stream4);
        // hipStreamSynchronize(stream5);
        // hipStreamSynchronize(stream6);
        // hipStreamSynchronize(stream7);
        // for (int i = 0; i < 256; i++)  {
        //     t_hist[i] = t0_hist[i] + t1_hist[i] + t2_hist[i] + t3_hist[i] + t4_hist[i] + t5_hist[i] + t6_hist[i] + t7_hist[i];
        // }
    }
    hipDeviceSynchronize();

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("KapCUDA kernel launch error: %s\n", hipGetErrorString(err));
    }
    
    //histogramGPU<<<((gpuResult.h*gpuResult.w)/256)+1, 256, 256*sizeof(int) >>>(d_hist, d_ImgIn, gpuResult.w, gpuResult.h);

    err = hipMemcpy(t_hist, d_hist, 256 * sizeof(int), hipMemcpyDeviceToHost);  // Copy data from host to device

    hipEventRecord(stopCuda);
    hipEventSynchronize(stopCuda);
    hipEventElapsedTime(&millisecondsTransfers, startCuda, stopCuda);

    printf("\nGPU1 Execution time: %lf seconds\n", millisecondsTransfers/1000.0);
    time = histogram_equalization_prep(gpuResult.img, img_in.img, t_hist, gpuResult.w, gpuResult.h, 256, d_ImgIn);

    time += millisecondsTransfers;

    hipEventRecord(startCuda, 0);

    err = hipMemcpy(result.img, gpuResult.img, gpuResult.w * gpuResult.h * sizeof(unsigned char), hipMemcpyDeviceToHost);  // Copy data from host to device

    hipFree(d_ImgIn);  
    hipFree(d_hist);
    hipFree(gpuResult.img);

    hipEventRecord(stopCuda, 0);
    hipEventSynchronize(stopCuda);
    hipEventElapsedTime(&millisecondsTransfers, startCuda, stopCuda);

    time += millisecondsTransfers;

    printf("\nGPU Execution time: %lf seconds\n", time/1000.0);

    return result;
}