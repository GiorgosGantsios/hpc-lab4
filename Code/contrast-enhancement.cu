#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include "hist-equ.h"

PGM_IMG contrast_enhancement_GPU(PGM_IMG img_in)  {
    hipEvent_t startCuda, stopCuda;
    PGM_IMG gpuResult;
    PGM_IMG result;
    float millisecondsTransfers = 0, time;
    int t_hist[256];
    int *d_hist;
    unsigned char * d_ImgIn;

    result.w = img_in.w;
    result.h = img_in.h;
    
    hipMallocManaged(&gpuResult.img, result.w * result.h * sizeof(unsigned char));

    gpuResult.w = img_in.w;
    gpuResult.h = img_in.h;

    hipEventCreate(&startCuda);
    hipEventCreate(&stopCuda);

    hipEventRecord(startCuda);

    hipError_t err = hipMalloc((void**)&d_hist, 256 * sizeof(int));  // Allocate memory on the GPU
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA malloc error: %s\n", hipGetErrorString(err));
        if (gpuResult.img) hipFree(gpuResult.img);
        gpuResult.w = -1;
        return(gpuResult);
    }
    
    hipMemset(d_hist, 0, sizeof(int) * 256);
    
    histogramGPU<<<((gpuResult.h*gpuResult.w)/256)+1, 256, 256*sizeof(int) >>>(d_hist, img_in.img, gpuResult.w, gpuResult.h);

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
        if (gpuResult.img) hipFree(gpuResult.img);
        if (d_hist) hipFree(d_hist);
        gpuResult.w = -1;
        return(gpuResult);
    }

    err = hipMemcpy(t_hist, d_hist, 256 * sizeof(int), hipMemcpyDeviceToHost);  // Copy data from host to device
    if (err != hipSuccess) {
        printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
        if (gpuResult.img) hipFree(gpuResult.img);
        if (d_hist) hipFree(d_hist);
        gpuResult.w = -1;
        return(gpuResult);
    }

    hipEventRecord(stopCuda);
    hipEventSynchronize(stopCuda);
    hipEventElapsedTime(&millisecondsTransfers, startCuda, stopCuda);

    time = histogram_equalization_prep(gpuResult.img, img_in.img, t_hist, gpuResult.w, gpuResult.h, 256, img_in.img);

    if (time == -1)  {
        if (gpuResult.img) hipFree(gpuResult.img);
        if (d_hist) hipFree(d_hist);
        gpuResult.w = -1;
        return(gpuResult);
    }

    time += millisecondsTransfers;

    hipEventRecord(startCuda, 0);

    hipFree(d_hist);

    hipEventRecord(stopCuda, 0);
    hipEventSynchronize(stopCuda);
    hipEventElapsedTime(&millisecondsTransfers, startCuda, stopCuda);

    time += millisecondsTransfers;

    printf("\nGPU Execution time: %lf seconds\n", time/1000.0);

    return gpuResult;
}