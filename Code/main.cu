#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include "hist-equ.h"

bool run_GPU_gray_test(PGM_IMG img_in, char *out_filename);

int main(int argc, char *argv[]){
    PGM_IMG img_ibuf_g;
    bool result;

	if (argc != 3) {
		printf("Run with input file name and output file name as arguments\n");
		exit(1);
	}
	
    printf("Running contrast enhancement for gray-scale images.\n");
    img_ibuf_g = read_pgm(argv[1]);

    result = run_GPU_gray_test(img_ibuf_g, argv[2]);

    if (result == false)  {
        free_pgm(img_ibuf_g);
        return(0);
    }
    free_pgm(img_ibuf_g);

	return 0;
}

bool run_GPU_gray_test(PGM_IMG img_in, char *out_filename)
{
    PGM_IMG img_obuf;
    
    printf("Starting GPU processing...\n");
    img_obuf = contrast_enhancement_GPU(img_in);
    if (img_obuf.w == -1)  {
        free_pgm(img_in);
        return(false);
    }
    write_pgm(img_obuf, out_filename);
    free_pgm(img_obuf);
    return(true);
}

PGM_IMG read_pgm(const char * path){
    FILE * in_file;
    char sbuf[256];
    
    
    PGM_IMG result;
    int v_max;
    in_file = fopen(path, "r");
    if (in_file == NULL){
        printf("Input file not found!\n");
        exit(1);
    }
    
    fscanf(in_file, "%s", sbuf); /*Skip the magic number*/
    fscanf(in_file, "%d",&result.w);
    fscanf(in_file, "%d",&result.h);
    fscanf(in_file, "%d\n",&v_max);
    printf("Image size: %d x %d\n", result.w, result.h);
    
    hipMallocManaged(&result.img, result.w * result.h * sizeof(unsigned char));
        
    fread(result.img,sizeof(unsigned char), result.w*result.h, in_file);    
    fclose(in_file);
    
    return result;
}

void write_pgm(PGM_IMG img, const char * path){
    FILE * out_file;
    out_file = fopen(path, "wb");
    fprintf(out_file, "P5\n");
    fprintf(out_file, "%d %d\n255\n",img.w, img.h);
    fwrite(img.img,sizeof(unsigned char), img.w*img.h, out_file);
    fclose(out_file);
}

void free_pgm(PGM_IMG img)
{
    hipFree(img.img);
}